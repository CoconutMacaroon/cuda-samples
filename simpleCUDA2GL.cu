#include "hip/hip_runtime.h"
#include "simpleCUDA2GL.h"

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__device__ double dot(const double x[3], const double y[3]) {
    return (x[0] * y[0]) + (x[1] * y[1]) + (x[2] * y[2]);
}

__device__ void add(const double a[], const double b[], double *resultLocation) {
    resultLocation[0] = a[0] + b[0];
    resultLocation[1] = a[1] + b[1];
    resultLocation[2] = a[2] + b[2];
}

__device__ void subtract(const double a[], const double b[], double *resultLocation) {
    resultLocation[0] = a[0] - b[0];
    resultLocation[1] = a[1] - b[1];
    resultLocation[2] = a[2] - b[2];
}

__device__ void multiply(double a, const double b[], double *resultLocation) {
    resultLocation[0] = a * b[0];
    resultLocation[1] = a * b[1];
    resultLocation[2] = a * b[2];
}

__device__ void canvasToViewport(int x, int y, double *returnLocation) {
    returnLocation[0] = x * VIEWPORT_WIDTH / (double) CANVAS_WIDTH;
    returnLocation[1] = y * VIEWPORT_HEIGHT / (double) CANVAS_HEIGHT;
    returnLocation[2] = D;
}

__device__ void reflectRay(double R[], double N[], double *returnLocation) {
    double n_dot_r = dot(N, R);
    double n_multiply_two[3];
    multiply(2, N, n_multiply_two);

    double dot_times_multiply[3];
    multiply(n_dot_r, n_multiply_two, dot_times_multiply);

    subtract(dot_times_multiply, R, returnLocation);
}

__device__ void intersectRaySphere(double cameraPos[], double d[], Sphere sphere, double *returnLocation) {
    double CO[3];
    subtract(cameraPos, sphere.center, CO);

    double a = dot(d, d);
    double b = 2 * dot(CO, d);
    double c = dot(CO, CO) - sphere.radius * sphere.radius;

    double discriminant = b * b - 4 * a * c;

    if (discriminant < 0) {
        returnLocation[0] = inf;
        returnLocation[1] = inf;
        return;
    }

    double discriminantSqrt = sqrt(discriminant);

    returnLocation[0] = (double) ((-b + discriminantSqrt) / (2 * a));
    returnLocation[1] = (double) ((-b - discriminantSqrt) / (2 * a));
}

__device__ IntersectionData closestIntersection(double cameraPos[], double d[], double t_min, double t_max) {
    double closest_t = inf;
    Sphere closestSphere;
    bool isNull = true;
    for (size_t i = 0; i < ARR_LEN(spheres); ++i) {
        double t[2];
        intersectRaySphere(cameraPos, d, spheres[i], t);

        if (t[0] < closest_t && t_min < t[0] && t[0] < t_max) {
            closest_t = t[0];
            closestSphere = spheres[i];
            isNull = false;
        }
        if (t[1] < closest_t && t_min < t[1] && t[1] < t_max) {
            closest_t = t[1];
            closestSphere = spheres[i];
            isNull = false;
        }
    }
    IntersectionData data = {.sphere = closestSphere, .closest_t = closest_t, .isSphereNull = isNull};
    return data;
}

__device__ double computeLighting(double P[], double N[], double V[], double s) {
    double intensity = 0.0;
    for (size_t i = 0; i < ARR_LEN(lights); ++i) {
        if (lights[i].lightType == LIGHT_TYPE_AMBIENT) {
            intensity += lights[i].intensity;
        } else {
            double L[3];
            double t_max;
            if (lights[i].lightType == LIGHT_TYPE_POINT) {
                subtract(lights[i].position, P, L);
                t_max = 1.0;
            } else {
                L[0] = lights[i].direction[0];
                L[1] = lights[i].direction[1];
                L[2] = lights[i].direction[2];
                t_max = DBL_MAX;
            }
            // shadow check
            IntersectionData intersectionData = closestIntersection(P, L, 0.001, t_max);

            if (!intersectionData.isSphereNull)
                continue;

            // diffuse
            double n_dot_l = dot(N, L);

            if (n_dot_l > 0)
                intensity += lights[i].intensity * n_dot_l / (LENGTH(N) * LENGTH(L));

            // specular
            if (s != -1) {
                // 2 * N * dot(N, L) - L
                double R[3];

                reflectRay(L, N, R);

                double r_dot_v = dot(R, V);

                if (r_dot_v > 0)
                    intensity += lights[i].intensity * pow(r_dot_v / (LENGTH(R) * LENGTH(V)), s);
            }
        }
    }
    return intensity;
}

__device__ Color traceRay(double cameraPos[3], double d[], double min_t, double max_t, int recursion_depth) {
    IntersectionData intersectionData = closestIntersection(cameraPos, d, min_t, max_t);
    if (intersectionData.isSphereNull)
        return BACKGROUND_COLOR;

    double tmp1[3];
    multiply(intersectionData.closest_t, d, tmp1);

    double P[3];
    add(cameraPos, tmp1, P);

    double N[3];
    subtract(P, intersectionData.sphere.center, N);

    double N2[3];
    multiply(1.0 / LENGTH(N), N, N2);

    double tmp3[3];
    multiply(-1.0, d, tmp3);
    double lighting = computeLighting(P, N, tmp3, intersectionData.sphere.specular);
    Color localColor = {ROUND_COLOR(intersectionData.sphere.color.r * lighting),
                        ROUND_COLOR(intersectionData.sphere.color.g * lighting),
                        ROUND_COLOR(intersectionData.sphere.color.b * lighting)};

    if (recursion_depth <= 0 || intersectionData.sphere.reflectiveness <= 0)
        return localColor;

    double temp[3];
    multiply(-1.0, d, temp);
    double R[3];
    reflectRay(temp, N2, R);

    Color reflectedColor = traceRay(P, R, 0.001, inf, recursion_depth - 1);
    return (Color) {ROUND_COLOR(localColor.r * (1 - intersectionData.sphere.reflectiveness) +
                                reflectedColor.r * intersectionData.sphere.reflectiveness), ROUND_COLOR(
                            localColor.g * (1 - intersectionData.sphere.reflectiveness) +
                            reflectedColor.g * intersectionData.sphere.reflectiveness), ROUND_COLOR(
                            localColor.b * (1 - intersectionData.sphere.reflectiveness) +
                            reflectedColor.b * intersectionData.sphere.reflectiveness)};
}

__global__ void cudaProcess(unsigned int *g_odata, int imgw) {
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;
    /*
     * THIS IS HOW WE DRAW A PIXEL:
     * g_odata[y * imgw + x] = rgbToInt(0, 255, 255);
     */

    double d[3];
    canvasToViewport(x - (CANVAS_WIDTH / 2), y - (CANVAS_HEIGHT / 2), d);
    Color c = traceRay(cameraPosition, d, 1, inf, RECURSION_DEPTH_FOR_REFLECTIONS);
    g_odata[y * imgw + x] = rgbToInt(c.r, c.g, c.b);
}

__global__ void moveCamera(double z, double y, double x) {
    cameraPosition[0] += x;
    cameraPosition[1] += y;
    cameraPosition[2] += z;
}
extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw) {
    keyActions();
    cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw);
}

extern void moveCam(double z, double y, double x) {
    moveCamera<<<1, 1>>>(z, y, x);
}
